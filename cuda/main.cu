
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

const size_t TILE_WIDTH = 16; // block width == tile width

__global__ void matmul(float* Md, float* Nd, float* outd, int width) {
	__shared__ float Mtile[TILE_WIDTH][TILE_WIDTH]; // 1 shared tile per block
	__shared__ float Ntile[TILE_WIDTH][TILE_WIDTH];
	
	int bx = blockIdx.x;
	int by = blockIdx.y;
	
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	
	int row = by * TILE_WIDTH + ty; // row and column indices of resulting value
	int col = bx * TILE_WIDTH + tx;
	
	float outval = 0.0;
	for (int ntile = 0; ntile < width / TILE_WIDTH; ntile++) {
		// (ntile * TILE_WIDTH) elements have already been processed, so skip them
		Mtile[ty][tx] = Md[row * width + (ntile * TILE_WIDTH + tx)];
		Ntile[ty][tx] = Nd[(ntile * TILE_WIDTH + ty) * width + col];
		
		__syncthreads();
		
		for (int n = 0; n < TILE_WIDTH; n++) {
			outval += Mtile[ty][n] * Ntile[n][tx];
			__syncthreads();
		}
		outd[row * width + col] = outval;
	}
}

void printDeviceInfo(bool moreinfo) {
    int device;
    hipGetDevice(&device);

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device);
    
    int driver;
    hipDriverGetVersion(&driver);
    
    int runtime;
    hipRuntimeGetVersion(&runtime);
    
    cout << "GPU: " << prop.name << endl;
    cout << "Driver Version: " << driver << endl;
    cout << "Runtime Version: " << runtime << endl << endl;
	
	if (moreinfo) {
		cout << "warp size (threads): " << prop.warpSize << endl;
		cout << "global memory available (MiB): " << prop.totalGlobalMem / 1048576 << endl;
		cout << "shared memory available per block (bytes): " << prop.sharedMemPerBlock << endl;
		cout << "max threads per block: " << prop.maxThreadsPerBlock << endl;
		cout << "max number of blocks: (x: " << prop.maxGridSize[0] << ", y: " << prop.maxGridSize[1] << ", z: " << prop.maxGridSize[2] << ")" << endl << endl;
	}
}

int main(int argc, char **argv) {
	printDeviceInfo(true);
	
	const unsigned int width = 64;
	const unsigned int blockwidth = 16;
	
	float A[width * width] = { 0 };
	float B[width * width] = { 0 };
	
	for (int i = 0; i < width; i++) {
		A[width * i + i] = 1.0;
		B[width * i + i] = 1.0;
	}
	
	float C[width * width];
	
	float* Ad;
	float* Bd;
	float* Cd;
	
	const unsigned int size = width * width * sizeof(float);
	
	hipMalloc((void**) &Ad, size);
	hipMalloc((void**) &Bd, size);
	hipMalloc((void**) &Cd, size);

	hipMemcpy(Ad, A, size, hipMemcpyHostToDevice);
	hipMemcpy(Bd, B, size, hipMemcpyHostToDevice);
	
	dim3 dimBlock(blockwidth, blockwidth, 1);
	dim3 dimGrid(width / blockwidth, width / blockwidth, 1); // dim3 = vec3, 1 for final element because grids have to be 2D
	
	matmul<<<dimGrid, dimBlock>>>(Ad, Bd, Cd, width);
	
	hipMemcpy(C, Cd, size, hipMemcpyDeviceToHost);
	
	for (size_t i = 0; i < width * width; i++) {
		if ((i % width == 0) && (i > 0)) {
			cout << endl;
		}
		cout << C[i] << " ";
	}
	cout << endl;
	
	hipFree(Ad);
	hipFree(Bd);
	hipFree(Cd);
}
